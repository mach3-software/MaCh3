#include "hip/hip_runtime.h"
#include "mcmc/gpuMCMCProcessorUtils.cuh"

// ******************************************
// CONSTANTS
// ******************************************

// d_NAME declares DEVICE constants (live on GPU)
__device__ __constant__ int d_nLag;
__device__ __constant__ int d_nDraws;
__device__ __constant__ int d_nEntries;

// h_NAME declares HOST constants (live on CPU)
static int h_nLag     = -1;
static int h_nDraws   = -1;
static int h_nEntries = -1;

// *******************************************
//              INITIALISE GPU
// *******************************************

// *******************************************
/// KS: Initialiser, here we allocate memory for variables and copy constants
__host__ void InitGPU_AutoCorr(
// *******************************************
                          float **ParStep_gpu,
                          float **NumeratorSum_gpu,
                          float **ParamSums_gpu,
                          float **DenomSum_gpu,

                          int n_Entries,
                          int n_Pars,
                          const int n_Lags) {

  // Write to the global statics (h_* denotes host stored variable)
  h_nDraws = n_Pars;
  h_nLag = n_Lags;
  h_nEntries = n_Entries;

  // Copy the constants
  hipMemcpyToSymbol(HIP_SYMBOL(d_nLag),   &h_nLag,   sizeof(h_nLag));
  CudaCheckError();

  hipMemcpyToSymbol(HIP_SYMBOL(d_nDraws), &h_nDraws, sizeof(h_nDraws));
  CudaCheckError();

  hipMemcpyToSymbol(HIP_SYMBOL(d_nEntries), &h_nEntries, sizeof(h_nEntries));
  CudaCheckError();

  // Allocate chunks of memory to GPU
  //Numerator which is directly used for calculating LagL
  hipMalloc((void **) NumeratorSum_gpu, h_nLag*h_nDraws*sizeof(float));
  CudaCheckError();

  //Denominator which is directly used for calculating LagL
  hipMalloc((void **) DenomSum_gpu, h_nLag*h_nDraws*sizeof(float));
  CudaCheckError();

  //Mean value for a given parameter
  hipMalloc((void **) ParamSums_gpu, h_nDraws*sizeof(float));
  CudaCheckError();

  //store value of paramter for each step
  hipMalloc((void **) ParStep_gpu, h_nDraws*h_nEntries*sizeof(float*));
  CudaCheckError();

  printf(" Allocated in total %f MB for autocorrelations calculations on GPU\n", double(sizeof(float)*(h_nLag*h_nDraws+h_nLag*h_nDraws+h_nDraws+h_nDraws*h_nEntries))/1.E6);

}

// ******************************************************
//                START COPY TO GPU
// ******************************************************

// ******************************************************
/// KS: Copy necessary variables from CPU to GPU
__host__ void CopyToGPU_AutoCorr(
// ******************************************************
                            float *ParStep_cpu,
                            float *NumeratorSum_cpu,
                            float *ParamSums_cpu,
                            float *DenomSum_cpu,

                            float *ParStep_gpu,
                            float *NumeratorSum_gpu,
                            float *ParamSums_gpu,
                            float *DenomSum_gpu) {

  //store value of parameter for each step
  hipMemcpy(ParStep_gpu, ParStep_cpu, h_nDraws*h_nEntries*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();

  //Mean value for a given parameter
  hipMemcpy(ParamSums_gpu, ParamSums_cpu, h_nDraws*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();

  //Numerator which is directly used for calculating LagL
  hipMemcpy(NumeratorSum_gpu, NumeratorSum_cpu, h_nLag*h_nDraws*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();

  //Denominator which is directly used for calculating LagL
  hipMemcpy(DenomSum_gpu, DenomSum_cpu, h_nLag*h_nDraws*sizeof(float), hipMemcpyHostToDevice);
  CudaCheckError();
}


// ********************************************************
//                  START GPU KERNELS
//*********************************************************

//*********************************************************
/// Eval autocorrelations based on Box and Jenkins
__global__ void EvalOnGPU_AutoCorr(
    const float* __restrict__ ParStep_gpu,
    const float* __restrict__ ParamSums_gpu,
    float*  NumeratorSum_gpu,
    float*  DenomSum_gpu) {
//*********************************************************

  const unsigned int CurrentLagNum = (blockIdx.x * blockDim.x + threadIdx.x);

  //KS: Accessing shared memory is much much faster than global memory hence we use shared memory for calculation and then write to global memory
  __shared__ float shared_NumeratorSum[_BlockSize_];
  __shared__ float shared_DenomSum[_BlockSize_];

  // this is the stopping condition!
  if (CurrentLagNum < d_nLag*d_nDraws)
  {
      shared_NumeratorSum[threadIdx.x] = 0;
      shared_DenomSum[threadIdx.x] = 0;

      //KS: Might consider caching this information, which MIGHT be faster too lazy right now
      const int Param = int(CurrentLagNum/d_nLag);
      const int nLag = CurrentLagNum - Param*d_nLag;
      // Loop over the number of entries
      for (int i = 0; i < d_nEntries; ++i)
      {
        //KS: Use fmaf to have it tiny bit faster, for something easier to read: Param*d_nEntries + i
        int CurrParStep = fmaf(Param, d_nEntries, i);
        const float Diff = ParStep_gpu[CurrParStep]-ParamSums_gpu[Param];
        // Only sum the numerator up to i = N-k
        if (i < d_nEntries-nLag)
        {
          //KS: Use fmaf to have it tiny bit faster, for something easier to read: Param*d_nEntries + (i + nLag)
          CurrParStep = fmaf(Param, d_nEntries, i + nLag);
          const float LagTerm = ParStep_gpu[CurrParStep]-ParamSums_gpu[Param];
          const float Product = Diff*LagTerm;
          shared_NumeratorSum[threadIdx.x] += Product;
        }
        // Square the difference to form the denominator
        const float Denom = Diff*Diff;
        shared_DenomSum[threadIdx.x] += Denom;
      }

      //KS: Make sure threads are synchronised before moving to global memory
      __syncthreads();
      NumeratorSum_gpu[CurrentLagNum] = shared_NumeratorSum[threadIdx.x];
      DenomSum_gpu[CurrentLagNum]     = shared_DenomSum[threadIdx.x];
  }
}

// *****************************************
/// KS: This call the main kernel responsible for calculating LagL and later copy results back to CPU
__host__ void RunGPU_AutoCorr(
    float*  ParStep_gpu,
    float*  ParamSums_gpu,
    float*  NumeratorSum_gpu,
    float*  DenomSum_gpu,
    float*  NumeratorSum_cpu,
    float*  DenomSum_cpu) {
// *****************************************

  dim3 block_size;
  dim3 grid_size;

  block_size.x = _BlockSize_;
  grid_size.x = (h_nLag*h_nDraws / block_size.x) + 1;

  EvalOnGPU_AutoCorr<<<grid_size, block_size>>>(
      ParStep_gpu,
      ParamSums_gpu,
      NumeratorSum_gpu,
      DenomSum_gpu);
  CudaCheckError();

  printf(" Finished calculating now copying results back to CPU \n");

  //KS: Finally copy paste memory from GPU to CPU
  hipMemcpy(NumeratorSum_cpu, NumeratorSum_gpu, h_nLag*h_nDraws*sizeof(float), hipMemcpyDeviceToHost);
  CudaCheckError();

  hipMemcpy(DenomSum_cpu, DenomSum_gpu, h_nLag*h_nDraws*sizeof(float), hipMemcpyDeviceToHost);
  CudaCheckError();
}

// *********************************
// CLEANING
// *********************************

// *********************************
/// KS: free memory on gpu
__host__ void CleanupGPU_AutoCorr(
    float *ParStep_gpu,
    float *NumeratorSum_gpu,
    float *ParamSums_gpu,
    float *DenomSum_gpu) {
// *********************************
  hipFree(ParStep_gpu);
  hipFree(NumeratorSum_gpu);
  hipFree(ParamSums_gpu);
  hipFree(DenomSum_gpu);

  printf(" Cleared memory at GPU, I am free \n");
  return;
}
